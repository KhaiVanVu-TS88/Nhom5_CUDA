#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <hip/hip_runtime.h>

#define N 2000

using namespace std;

__global__ void matrixMultiplyKernel(int* A, int* B, int* C, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < n) {
        int sum = 0;
        for (int k = 0; k < n; k++) {
            sum += A[row * n + k] * B[k * n + col];
        }
        C[row * n + col] = sum;
    }
}

int main() {
    
    int* A = new int[N * N];
    int* B = new int[N * N];
    int* C = new int[N * N];
    // Khởi tạo ma trận A và B gồm các phần tử đều là 1
    for (int i = 0; i < N * N; ++i) {
        A[i] = 1;
        B[i] = 1;
        C[i] = 0; // Khởi tạo ma trận C với 0
    }

    int* d_A, * d_B, * d_C;

    // Cấp phát bộ nhớ trên GPU
    hipMalloc((void**)&d_A, N * N * sizeof(int));
    hipMalloc((void**)&d_B, N * N * sizeof(int));
    hipMalloc((void**)&d_C, N * N * sizeof(int));

    // Copy dữ liệu từ CPU sang GPU
    hipMemcpy(d_A, A, N * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N * N * sizeof(int), hipMemcpyHostToDevice);

    // Thiết lập kích thước lưới và block
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x, (N + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // Khởi tạo các sự kiện để tính thời gian
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Bắt đầu tính thời gian
    hipEventRecord(start);

    // Gọi kernel để thực hiện phép nhân
    matrixMultiplyKernel << <numBlocks, threadsPerBlock >> > (d_A, d_B, d_C, N);

    // Dừng tính thời gian
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Tính toán thời gian
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Copy kết quả từ GPU về CPU
    hipMemcpy(C, d_C, N * N * sizeof(int), hipMemcpyDeviceToHost);

    // Thời gian chạy song song
    cout << "N = " << N << endl;
    cout << "Thoi gian chay: " << milliseconds << " ms" << endl;

    // Giải phóng bộ nhớ
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    delete[] A;
    delete[] B;
    delete[] C;

    // Giải phóng các sự kiện
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}